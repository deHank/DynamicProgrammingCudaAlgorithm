#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <cuco/dynamic_map.cuh>


// Calculates the distance between two instances
__device__ float generateItemSet(float* instance_A, float* instance_B, int num_attributes) {
    float sum = 0;
    
    for (int i = 0; i < num_attributes-1; i++) {
        float diff = instance_A[i] - instance_B[i];
        //printf("instance a and b were %.3f %.3f\n", instance_A[i] ,instance_B[i]);
        sum += diff*diff;
    }
    //printf("sum was %.3f\n,", sum);
    return sqrt(sum);
}

__global__ void processItemSets(char *inData, int minimumSetNum, int *d_Offsets, int totalRecords, cuco::dynamic_map<int, int>* hashmaps){
    //we know that tid will be the row
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int sharedArray[];
    int items[32]; // Assuming a maximum of 32 items per transaction
    int itemCount = 0;
    int number = 0;
    bool inNumber = false;

    //cuco::dynamic_map<int, int>& blockMap = hashmaps[blockIdx.x];

    extern __shared__ int sharedMemory[];
    extern __shared__ int offsetFromStart[]; //offset in shared memory, where each thread 


    if(tid < totalRecords){
        //printf("our offest is %d\n", d_Offsets[tid]);
        char* line = inData + d_Offsets[tid];
        //const char* current = line;
        
            //int maxSetSize = 0;
            // Parse the line to extract items
            for (char* current = line; *current != '\n' && *current != '\0'; current++) {
                if (*current >= '0' && *current <= '9') {
                    number = number * 10 + (*current - '0');
                    inNumber = true;
                } else if (inNumber) {
                    items[itemCount++] = number;
                    number = 0;
                    inNumber = false;
                }
            }
            if (inNumber) {
                items[itemCount++] = number;
            }


            // Generate all subsets
            int totalSubsets = 1 << itemCount; // 2^itemCount
            offsetFromStart[threadIdx.x] = pow(2, itemCount);
            __syncthreads();
            if(tid == 23640){
                int beginningOffset = 0;
                for (int mask = 0; mask < totalSubsets; mask++) {
                    int lengthOfKey = 0;
                    printf("{ ");
                    for (int i = 0; i < itemCount; i++) {
                        char* subSet;
                        if (mask & (1 << i)) { 
                            
                                printf("%d ", items[i]);
                            
                        }

                        //suppose I had the concatinated string here
                        
                        

                    }
                    printf("}\n");
                }
                for(int j = 0; j < threadIdx.x; j++){
                    beginningOffset = beginningOffset + offsetFromStart[j];
                }
                printf("we will begin to write into the shared memory at %d\n", beginningOffset);

            }
        if(tid == 23640){
            printf("block offset to tid %d is %d\n", threadIdx.x, offsetFromStart[threadIdx.x]);
        }
    }
}


__global__ void printStuff(float *test_matrix, float *train_matrix, 
int numElements, int train_num_instances, int k, int num_attributes, int num_classes, int *predictionsGlobal, int testNumInstances, int stream, int testNumInstancePerStream){
    
    //threadid within this block
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < testNumInstancePerStream){
        int idx = stream * testNumInstancePerStream + tid;
        int classCounts[10];  // Each thread gets its own copy of 'localArray'
        // Declare a single shared memory block
        
        // Assign parts of sharedMemory to different arrays
        float candidates[6];                           // First part: candidates
        //int* predictions = (int*)&sharedMemory;     // Second part: predictions
        //int* classCounts = (int*)&sharedMemory[2 * k * blockDim.x + testNumInstances]; // Third part: class counts

        for(int i = 0; i < 6; i++){ candidates[i] = FLT_MAX; }

        //__shared__ float candidates[k*2 * blockDim.x]; //we need to do k*2 * blocksize (blockDim.x)
        //__shared__ int predictions[numElements]; //predicitons array in shared memory (no penalty)
        
        
        
        

    
        //so i know this thread index is less than the number of test elements
        //meaning i would want to run the KNN algorithm for this section 
        //I could call this the outer loop of the KNN Algorithm
        //printf("test\n");
    
    
        //__shared__ float predictions[numElements];

        int queryIndex = idx;
        //and obviously we want to get stride to = 1 
        //But we can use shared memory to compensate for bad strides
        //so is the best way to share the entire training array
        //and that way each test point begins at 1 in the training array
        //stride can be 1?
        //for each thread,

        
        for(int keyIndex = 0; keyIndex < train_num_instances; keyIndex++) {
                float dist = 1; 
            //float dist = distance(&test_matrix[queryIndex*num_attributes], &train_matrix[keyIndex*num_attributes], num_attributes);
            
            if(idx == 41){
                //printf("getting the distance at idx = %d and it was %.3f and total train size is %d num attributes is %d key index is %d\n",idx, dist, testNumInstances, num_attributes, keyIndex);
                //printf("train num instances is %d\n", train_num_instances);
            }
            
            
            //printf("our distance was %.3f, num classes is %d\n", dist, num_classes);
            // Add to our candidates
            for(int c = 0; c < k; c++){
                if(0 < candidates[2*c]) {
                    // Found a new candidate
                    // Shift previous candidates down by one
                    for(int x = k-2; x >= c; x--) {
                        candidates[2*x+2] = candidates[2*x];
                        candidates[2*x+3] = candidates[2*x+1];
                    }
                    
                    // Set key vector as potential k NN
                    candidates[2*c ] = dist;
                    candidates[2*c+1 ] = train_matrix[keyIndex*num_attributes + num_attributes - 1]; // class value

                    break;
                }
            }
        }

        // Bincount the candidate labels and pick the most common
        for(int i = 0; i < k; i++) {
            classCounts[(int)candidates[2*i+1 ] ] += 1;
            //printf("class counts here are %d\n", classCounts[(int)candidates[2*i+1 + candidatesOffset] + classCountsOffset]);
        }
        
        int max_value = -1;
        int max_class = 0;
        for(int i = 0; i < num_classes; i++) {
            if(classCounts[i] > max_value) {
                max_value = classCounts[i ];
                max_class = i;
                if(idx < 10){
                    //printf("max class at 41 was %d and the classCountOfset is %d \n", max_class, classCountsOffset);
                    //printf("my thread is is %d\n", threadIdx.x);
                }
                //printf("class count at i= %d is %d\n", i, classCounts[i + classCountsOffset]);
            }
            
        }

        //printf("Thread %d: max_class = %d\n", idx, max_class);
        //printf("predictions %d is %d\n", idx, predictions[idx]);
        __syncthreads();
        
        
        
        predictionsGlobal[idx] = max_class;
    
        // Make prediction with 
        
        
        //printf("max class was %d\n, and predictions at %d was %d\n", max_class, idx, predictions[idx]);


        
        
        
        //printf("max class was %d\n, and predictionsGlobal at %d was %d\n", max_class, idx, predictionsGlobal[idx]);


    }
}

// Implements a threaded kNN where for each candidate query an in-place priority queue is maintained to identify the nearest neighbors
int KNN() {     
        int lineCountInDataset = 1692082;
    const char* inDataFilePath = "sortedDataBase.txt";

    FILE* file = fopen(inDataFilePath, "r");

    // Get the file size
    fseek(file, 0, SEEK_END);
    size_t file_size = ftell(file);
    rewind(file);

    char* h_buffer = (char*)malloc(file_size);
    fread(h_buffer, 1, file_size, file);
    

    // Count the number of lines and create offsets
    int* h_offsets = (int*)malloc((file_size + 1) * sizeof(int));
    int lineCount = 0;
    h_offsets[lineCount++] = 0; // First line starts at the beginning
    
    for (size_t i = 0; i < file_size; i++) {
        //printf("are we in size?");
        if (h_buffer[i] == '\n') {
            //printf("we are in the newline stuff");
            h_offsets[lineCount++] = i + 1; // Next line starts after '\n'
            
        }
    }



    // Allocate memory to hold the file contents
    char* h_text = (char*)malloc(file_size);

    // Read the file into the host buffer
    fread(h_text, 1, file_size, file);
    //fclose(file);

    // Allocate memory on the GPU
    char* d_text;
    int* d_offsets; 
    hipMalloc(&d_text, file_size);
    hipMalloc(&d_offsets, lineCountInDataset * sizeof(int));
    // Copy the file contents to the GPU
    hipMemcpy(d_text, h_buffer, file_size, hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, lineCountInDataset * sizeof(int), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = ((lineCountInDataset + threadsPerBlock) - 1) /  threadsPerBlock; //how do we know how many blocks we need to use?

    //1_692_082 lineCount of Sorted DataBase
    int minItemCount = 3; //setting the minimum # of items to be considered an itemset

    //here I would want to generate all itemsets
    // Define sentinel values
    constexpr int empty_key = -1;   // An invalid key value
    constexpr int empty_value = -1; // An invalid value
    constexpr int erased_key = -2;  // An erased key marker

    // Initial capacity of the map
    std::size_t initial_capacity = 1024;

    // Create the map
    cuco::dynamic_map<int, int> global_map(
        initial_capacity,                // Initial capacity
        cuco::empty_key{empty_key},      // Empty key sentinel
        cuco::empty_value{empty_value},  // Empty value sentinel
        cuco::erased_key{erased_key}     // Erased key sentinel
    );


    // Allocate memory for the hashmaps on the device
    cuco::dynamic_map<int, int>* d_hashmaps;
    int numHashmaps = blocksPerGrid;

    hipMalloc(&d_hashmaps, numHashmaps * sizeof(cuco::dynamic_map<int, int>));

    // Initialize hashmaps on the host
    for (int i = 0; i < numHashmaps; i++) {
        cuco::dynamic_map<int, int> h_map(
            initial_capacity,
            cuco::empty_key{empty_key},
            cuco::empty_value{empty_value},
            cuco::erased_key{erased_key}
        );

        // Copy the constructed hashmap to the device
        hipMemcpy(&d_hashmaps[i], &h_map, sizeof(cuco::dynamic_map<int, int>), hipMemcpyHostToDevice);
    }



    processItemSets<<<blocksPerGrid, threadsPerBlock>>>(d_text, minItemCount, d_offsets, lineCountInDataset, d_hashmaps);
    hipDeviceSynchronize();
    return 1;



    //so essentially my goal is to produce all of these itemssets 
    //i want to take a previous gpu solution and use NVIDIA's hardware based 
    //dynamic programming APIs 
    //we send the transactional database to the GPU
    
    //1. We want to have a hashtable (key, value pairs)
    //our keys will be our itsemset, and our value will be the count 
    //we will do this for each itsemset and transaction
    
    //each thread will then reduce the hashtable into the shared memory
    //maybe we should begin with a pure list?
    //


    // int* predictions = (int*)malloc(test->num_instances() * sizeof(int));

    // int num_classes = train->num_classes();
    // int num_attributes = train->num_attributes();
    // int train_num_instances = train->num_instances();

    // int test_num_instances = test->num_instances();
    // int testNumClasses = test->num_classes();
    // int testNumAttributes = test-> num_attributes();
    

    // float *d_testSet;
    // float *d_trainSet;

    // int *d_predictions;

    // int numElements = num_attributes * train_num_instances;
    // int testNumElements = testNumAttributes * test_num_instances;
    // // Pointers representing the dataset as a 2D matrix size num_instances x num_attributes
    // float *train_matrix = train->get_dataset_matrix();  // Use directly
    // float *test_matrix = test->get_dataset_matrix();    // Use directly

    
    // int numStreams = 4;

    // //need to cudamalloc to hold the training data
    // //CUDA MALLOC for the train array
    // hipMalloc(&d_trainSet, (numElements) * sizeof(float));
    // hipMalloc(&d_testSet,(testNumElements) *  sizeof(float));
    // hipMalloc(&d_predictions, (test_num_instances * sizeof(int)));

    // hipStream_t *streams = (hipStream_t*) malloc (numStreams * sizeof(hipStream_t));

    // for (int i = 0; i < numStreams; i++){
    //     hipStreamCreate(&streams[i]);
    // }

    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     printf("CUDA error: %s\n", hipGetErrorString(err));
    // }

    // //int numberTrainElementsPerStream = (numElements + numStreams - 1) / numStreams;
    // int numberTestElementsPerStream = (testNumElements + numStreams - 1)/numStreams;
    // int numTestInstancesPerStream = (test_num_instances + numStreams - 1)/numStreams;
    
    
    
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // float milliseconds = 0;
    
    // int threadsPerBlockDim = 64;

    // // Calculate the size of shared memory needed
    // int sharedMemSize = 0;                  // For classCounts
    // printf("size of memory usage is %d\n", sharedMemSize);
    // int gridDimSize = (numTestInstancesPerStream + threadsPerBlockDim - 1) / threadsPerBlockDim;

    // hipEventRecord(start);
    // printf("before we call the kernel\n");
    // //hipMemcpy(d_trainSet, train_matrix, (numElements)*sizeof(float), hipMemcpyHostToDevice);
    // printf("size of numberTestElementsPerStream is %d\n", numberTestElementsPerStream);
    // printf("size of numberTestInstancePerStream is %d\n", numTestInstancesPerStream);
    // for (int i = 0; i < numStreams; i++)
    // {
    //     //copying the train set to our device (GPU)
    //     //0.041824 ms each way for 2 things... so why is performance with streams so poor?
    //     hipMemcpyAsync(d_trainSet, train_matrix, (numElements)*sizeof(float),hipMemcpyHostToDevice, streams[i]);
    //     hipMemcpyAsync(&d_testSet[i*numberTestElementsPerStream], &test_matrix[i*numberTestElementsPerStream], numberTestElementsPerStream  *  sizeof(float), hipMemcpyHostToDevice, streams[i]);
    //     //hipMemcpyAsync(&d_predictions[i*numTestInstancesPerStream], &predictions[i*numTestInstancesPerStream], numTestInstancesPerStream * sizeof(int), hipMemcpyHostToDevice, streams[i]);

    //     printStuff<<<gridDimSize, threadsPerBlockDim, sharedMemSize, streams[i]>>>(d_testSet, d_trainSet, numberTestElementsPerStream, train_num_instances, k, num_attributes, num_classes, d_predictions, test_num_instances, i, numTestInstancesPerStream);
        
    //     hipMemcpyAsync(&predictions[i*numTestInstancesPerStream], &d_predictions[i*numTestInstancesPerStream], numTestInstancesPerStream * sizeof(int), hipMemcpyDeviceToHost, streams[i]);

    // }

    // hipDeviceSynchronize();  // Ensure kernel finishes THIS IS SO IMPORTANT 
    // //OTHERWISE THE PROGRAM WILL JUST END BEFORE THE KERNELS EVEN GET A CHANCE TO LAUNCH

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("v1 GPU option time to sum the matrixes %f ms\n", milliseconds);

    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp, 0);

    // printf("GPU %d: %s\n", 0, deviceProp.name);
    
    

    // printf("after we call the kernel\n");
    
    // hipFree(d_trainSet);
    // hipFree(d_testSet);
    // free(streams);
    // hipFree(d_predictions);

    

    // return predictions;
}

int main(int argc, char *argv[])
{
    

    int x = KNN();
    return -1;  
}

/*  // Example to print the test dataset
    float* test_matrix = test->get_dataset_matrix();
    for(int i = 0; i < test->num_instances(); i++) {
        for(int j = 0; j < test->num_attributes(); j++)
            printf("%.0f, ", test_matrix[i*test->num_attributes() + j]);
        printf("\n");
    }
*/