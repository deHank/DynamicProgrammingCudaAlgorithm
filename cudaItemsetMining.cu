#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <cuco/dynamic_map.cuh>


// Calculates the distance between two instances
__device__ float generateItemSet(float* instance_A, float* instance_B, int num_attributes) {
    float sum = 0;
    
    for (int i = 0; i < num_attributes-1; i++) {
        float diff = instance_A[i] - instance_B[i];
        //printf("instance a and b were %.3f %.3f\n", instance_A[i] ,instance_B[i]);
        sum += diff*diff;
    }
    //printf("sum was %.3f\n,", sum);
    return sqrt(sum);
}

__global__ void processItemSets(char *inData, int minimumSetNum, int *d_Offsets, int totalRecords){
    //we know that tid will be the row
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int sharedArray[];
    int items[32]; // Assuming a maximum of 32 items per transaction
    int itemCount = 0;
    int number = 0;
    bool inNumber = false;

    if(tid < totalRecords){
        //printf("our offest is %d\n", d_Offsets[tid]);
        char* line = inData + d_Offsets[tid];
        //const char* current = line;
        if(tid == 23645){
            //int maxSetSize = 0;
            // Parse the line to extract items
            for (char* current = line; *current != '\n' && *current != '\0'; current++) {
                if (*current >= '0' && *current <= '9') {
                    number = number * 10 + (*current - '0');
                    inNumber = true;
                } else if (inNumber) {
                    items[itemCount++] = number;
                    number = 0;
                    inNumber = false;
                }
            }
            if (inNumber) {
                items[itemCount++] = number;
            }
            // Generate all subsets
            int totalSubsets = 1 << itemCount; // 2^itemCount
            for (int mask = 0; mask < totalSubsets; mask++) {
                printf("{ ");
                for (int i = 0; i < itemCount; i++) {
                    char* subSet;
                    if (mask & (1 << i)) { 
                        printf("%d ", items[i]);
                    }
                }
                printf("}\n");
            }
            //syncThreads();
        }
    }
}


__global__ void printStuff(float *test_matrix, float *train_matrix, 
int numElements, int train_num_instances, int k, int num_attributes, int num_classes, int *predictionsGlobal, int testNumInstances, int stream, int testNumInstancePerStream){
    
    //threadid within this block
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < testNumInstancePerStream){
        int idx = stream * testNumInstancePerStream + tid;
        int classCounts[10];  // Each thread gets its own copy of 'localArray'
        // Declare a single shared memory block
        
        // Assign parts of sharedMemory to different arrays
        float candidates[6];                           // First part: candidates
        //int* predictions = (int*)&sharedMemory;     // Second part: predictions
        //int* classCounts = (int*)&sharedMemory[2 * k * blockDim.x + testNumInstances]; // Third part: class counts

        for(int i = 0; i < 6; i++){ candidates[i] = FLT_MAX; }

        //__shared__ float candidates[k*2 * blockDim.x]; //we need to do k*2 * blocksize (blockDim.x)
        //__shared__ int predictions[numElements]; //predicitons array in shared memory (no penalty)
        
        
        
        

    
        //so i know this thread index is less than the number of test elements
        //meaning i would want to run the KNN algorithm for this section 
        //I could call this the outer loop of the KNN Algorithm
        //printf("test\n");
    
    
        //__shared__ float predictions[numElements];

        int queryIndex = idx;
        //and obviously we want to get stride to = 1 
        //But we can use shared memory to compensate for bad strides
        //so is the best way to share the entire training array
        //and that way each test point begins at 1 in the training array
        //stride can be 1?
        //for each thread,

        
        for(int keyIndex = 0; keyIndex < train_num_instances; keyIndex++) {
                float dist = 1; 
            //float dist = distance(&test_matrix[queryIndex*num_attributes], &train_matrix[keyIndex*num_attributes], num_attributes);
            
            if(idx == 41){
                //printf("getting the distance at idx = %d and it was %.3f and total train size is %d num attributes is %d key index is %d\n",idx, dist, testNumInstances, num_attributes, keyIndex);
                //printf("train num instances is %d\n", train_num_instances);
            }
            
            
            //printf("our distance was %.3f, num classes is %d\n", dist, num_classes);
            // Add to our candidates
            for(int c = 0; c < k; c++){
                if(0 < candidates[2*c]) {
                    // Found a new candidate
                    // Shift previous candidates down by one
                    for(int x = k-2; x >= c; x--) {
                        candidates[2*x+2] = candidates[2*x];
                        candidates[2*x+3] = candidates[2*x+1];
                    }
                    
                    // Set key vector as potential k NN
                    candidates[2*c ] = dist;
                    candidates[2*c+1 ] = train_matrix[keyIndex*num_attributes + num_attributes - 1]; // class value

                    break;
                }
            }
        }

        // Bincount the candidate labels and pick the most common
        for(int i = 0; i < k; i++) {
            classCounts[(int)candidates[2*i+1 ] ] += 1;
            //printf("class counts here are %d\n", classCounts[(int)candidates[2*i+1 + candidatesOffset] + classCountsOffset]);
        }
        
        int max_value = -1;
        int max_class = 0;
        for(int i = 0; i < num_classes; i++) {
            if(classCounts[i] > max_value) {
                max_value = classCounts[i ];
                max_class = i;
                if(idx < 10){
                    //printf("max class at 41 was %d and the classCountOfset is %d \n", max_class, classCountsOffset);
                    //printf("my thread is is %d\n", threadIdx.x);
                }
                //printf("class count at i= %d is %d\n", i, classCounts[i + classCountsOffset]);
            }
            
        }

        //printf("Thread %d: max_class = %d\n", idx, max_class);
        //printf("predictions %d is %d\n", idx, predictions[idx]);
        __syncthreads();
        
        
        
        predictionsGlobal[idx] = max_class;
    
        // Make prediction with 
        
        
        //printf("max class was %d\n, and predictions at %d was %d\n", max_class, idx, predictions[idx]);


        
        
        
        //printf("max class was %d\n, and predictionsGlobal at %d was %d\n", max_class, idx, predictionsGlobal[idx]);


    }
}

// Implements a threaded kNN where for each candidate query an in-place priority queue is maintained to identify the nearest neighbors
int KNN() {     
        int lineCountInDataset = 1692082;
    const char* inDataFilePath = "sortedDataBase.txt";

    FILE* file = fopen(inDataFilePath, "r");

    // Get the file size
    fseek(file, 0, SEEK_END);
    size_t file_size = ftell(file);
    rewind(file);

    char* h_buffer = (char*)malloc(file_size);
    fread(h_buffer, 1, file_size, file);
    

    // Count the number of lines and create offsets
    int* h_offsets = (int*)malloc((file_size + 1) * sizeof(int));
    int lineCount = 0;
    h_offsets[lineCount++] = 0; // First line starts at the beginning
    
    for (size_t i = 0; i < file_size; i++) {
        //printf("are we in size?");
        if (h_buffer[i] == '\n') {
            //printf("we are in the newline stuff");
            h_offsets[lineCount++] = i + 1; // Next line starts after '\n'
            
        }
    }



    // Allocate memory to hold the file contents
    char* h_text = (char*)malloc(file_size);

    // Read the file into the host buffer
    fread(h_text, 1, file_size, file);
    //fclose(file);

    // Allocate memory on the GPU
    char* d_text;
    int* d_offsets; 
    hipMalloc(&d_text, file_size);
    hipMalloc(&d_offsets, lineCountInDataset * sizeof(int));
    // Copy the file contents to the GPU
    hipMemcpy(d_text, h_buffer, file_size, hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, lineCountInDataset * sizeof(int), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = ((lineCountInDataset + threadsPerBlock) - 1) /  threadsPerBlock; //how do we know how many blocks we need to use?

    //1_692_082 lineCount of Sorted DataBase
    int minItemCount = 3; //setting the minimum # of items to be considered an itemset

    //here I would want to generate all itemsets

    processItemSets<<<blocksPerGrid, threadsPerBlock>>>(d_text, minItemCount, d_offsets, lineCountInDataset);
    hipDeviceSynchronize();
    return 1;



    //so essentially my goal is to produce all of these itemssets 
    //i want to take a previous gpu solution and use NVIDIA's hardware based 
    //dynamic programming APIs 
    //we send the transactional database to the GPU
    
    //1. We want to have a hashtable (key, value pairs)
    //our keys will be our itsemset, and our value will be the count 
    //we will do this for each itsemset and transaction
    
    //each thread will then reduce the hashtable into the shared memory
    //maybe we should begin with a pure list?
    //


    // int* predictions = (int*)malloc(test->num_instances() * sizeof(int));

    // int num_classes = train->num_classes();
    // int num_attributes = train->num_attributes();
    // int train_num_instances = train->num_instances();

    // int test_num_instances = test->num_instances();
    // int testNumClasses = test->num_classes();
    // int testNumAttributes = test-> num_attributes();
    

    // float *d_testSet;
    // float *d_trainSet;

    // int *d_predictions;

    // int numElements = num_attributes * train_num_instances;
    // int testNumElements = testNumAttributes * test_num_instances;
    // // Pointers representing the dataset as a 2D matrix size num_instances x num_attributes
    // float *train_matrix = train->get_dataset_matrix();  // Use directly
    // float *test_matrix = test->get_dataset_matrix();    // Use directly

    
    // int numStreams = 4;

    // //need to cudamalloc to hold the training data
    // //CUDA MALLOC for the train array
    // hipMalloc(&d_trainSet, (numElements) * sizeof(float));
    // hipMalloc(&d_testSet,(testNumElements) *  sizeof(float));
    // hipMalloc(&d_predictions, (test_num_instances * sizeof(int)));

    // hipStream_t *streams = (hipStream_t*) malloc (numStreams * sizeof(hipStream_t));

    // for (int i = 0; i < numStreams; i++){
    //     hipStreamCreate(&streams[i]);
    // }

    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     printf("CUDA error: %s\n", hipGetErrorString(err));
    // }

    // //int numberTrainElementsPerStream = (numElements + numStreams - 1) / numStreams;
    // int numberTestElementsPerStream = (testNumElements + numStreams - 1)/numStreams;
    // int numTestInstancesPerStream = (test_num_instances + numStreams - 1)/numStreams;
    
    
    
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // float milliseconds = 0;
    
    // int threadsPerBlockDim = 64;

    // // Calculate the size of shared memory needed
    // int sharedMemSize = 0;                  // For classCounts
    // printf("size of memory usage is %d\n", sharedMemSize);
    // int gridDimSize = (numTestInstancesPerStream + threadsPerBlockDim - 1) / threadsPerBlockDim;

    // hipEventRecord(start);
    // printf("before we call the kernel\n");
    // //hipMemcpy(d_trainSet, train_matrix, (numElements)*sizeof(float), hipMemcpyHostToDevice);
    // printf("size of numberTestElementsPerStream is %d\n", numberTestElementsPerStream);
    // printf("size of numberTestInstancePerStream is %d\n", numTestInstancesPerStream);
    // for (int i = 0; i < numStreams; i++)
    // {
    //     //copying the train set to our device (GPU)
    //     //0.041824 ms each way for 2 things... so why is performance with streams so poor?
    //     hipMemcpyAsync(d_trainSet, train_matrix, (numElements)*sizeof(float),hipMemcpyHostToDevice, streams[i]);
    //     hipMemcpyAsync(&d_testSet[i*numberTestElementsPerStream], &test_matrix[i*numberTestElementsPerStream], numberTestElementsPerStream  *  sizeof(float), hipMemcpyHostToDevice, streams[i]);
    //     //hipMemcpyAsync(&d_predictions[i*numTestInstancesPerStream], &predictions[i*numTestInstancesPerStream], numTestInstancesPerStream * sizeof(int), hipMemcpyHostToDevice, streams[i]);

    //     printStuff<<<gridDimSize, threadsPerBlockDim, sharedMemSize, streams[i]>>>(d_testSet, d_trainSet, numberTestElementsPerStream, train_num_instances, k, num_attributes, num_classes, d_predictions, test_num_instances, i, numTestInstancesPerStream);
        
    //     hipMemcpyAsync(&predictions[i*numTestInstancesPerStream], &d_predictions[i*numTestInstancesPerStream], numTestInstancesPerStream * sizeof(int), hipMemcpyDeviceToHost, streams[i]);

    // }

    // hipDeviceSynchronize();  // Ensure kernel finishes THIS IS SO IMPORTANT 
    // //OTHERWISE THE PROGRAM WILL JUST END BEFORE THE KERNELS EVEN GET A CHANCE TO LAUNCH

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("v1 GPU option time to sum the matrixes %f ms\n", milliseconds);

    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp, 0);

    // printf("GPU %d: %s\n", 0, deviceProp.name);
    
    

    // printf("after we call the kernel\n");
    
    // hipFree(d_trainSet);
    // hipFree(d_testSet);
    // free(streams);
    // hipFree(d_predictions);

    

    // return predictions;
}

int main(int argc, char *argv[])
{
    

    int x = KNN();
    return -1;  
}

/*  // Example to print the test dataset
    float* test_matrix = test->get_dataset_matrix();
    for(int i = 0; i < test->num_instances(); i++) {
        for(int j = 0; j < test->num_attributes(); j++)
            printf("%.0f, ", test_matrix[i*test->num_attributes() + j]);
        printf("\n");
    }
*/