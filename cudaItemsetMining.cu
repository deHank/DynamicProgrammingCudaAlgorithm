#include "hip/hip_runtime.h"
    #include <stdio.h>
    #include <stdlib.h>
    #include <pthread.h>
    #include <bits/stdc++.h>
    #include <hip/hip_runtime.h>
    #include "libarff/arff_parser.h"
    #include "libarff/arff_data.h"


    struct ThreadData{
        ArffData *train;  // Pointer to the first array
        ArffData *test;  // Pointer to the second array
        int* predictions;    // Start index
        int TRstart;  //start index of array
        int TRend;      // End index
        int TTstart;
        int TTend;
        int k;  
    } ;



    // Calculates the distance between two instances
    __device__ float distance(float* instance_A, float* instance_B, int num_attributes) {
        float sum = 0;
        
        for (int i = 0; i < num_attributes-1; i++) {
            float diff = instance_A[i] - instance_B[i];
            //printf("instance a and b were %.3f %.3f\n", instance_A[i] ,instance_B[i]);
            sum += diff*diff;
        }
        //printf("sum was %.3f\n,", sum);
        return sqrt(sum);
    }



    __global__ void printStuff(float *test_matrix, float *train_matrix, 
    int numElements, int train_num_instances, int k, int num_attributes, int num_classes, int *predictionsGlobal, int testNumInstances, int stream, int testNumInstancePerStream){
        
        //threadid within this block
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < testNumInstancePerStream){
            int idx = stream * testNumInstancePerStream + tid;
            int classCounts[10];  // Each thread gets its own copy of 'localArray'
            // Declare a single shared memory block
            
            // Assign parts of sharedMemory to different arrays
            float candidates[6];                           // First part: candidates
            //int* predictions = (int*)&sharedMemory;     // Second part: predictions
            //int* classCounts = (int*)&sharedMemory[2 * k * blockDim.x + testNumInstances]; // Third part: class counts

            for(int i = 0; i < 6; i++){ candidates[i] = FLT_MAX; }

            //__shared__ float candidates[k*2 * blockDim.x]; //we need to do k*2 * blocksize (blockDim.x)
            //__shared__ int predictions[numElements]; //predicitons array in shared memory (no penalty)
            
          
            
            

        
            //so i know this thread index is less than the number of test elements
            //meaning i would want to run the KNN algorithm for this section 
            //I could call this the outer loop of the KNN Algorithm
            //printf("test\n");
        
        
            //__shared__ float predictions[numElements];

            int queryIndex = idx;
            //and obviously we want to get stride to = 1 
            //But we can use shared memory to compensate for bad strides
            //so is the best way to share the entire training array
            //and that way each test point begins at 1 in the training array
            //stride can be 1?
            //for each thread,

            
            for(int keyIndex = 0; keyIndex < train_num_instances; keyIndex++) {
                   
                float dist = distance(&test_matrix[queryIndex*num_attributes], &train_matrix[keyIndex*num_attributes], num_attributes);
                
                if(idx == 41){
                    //printf("getting the distance at idx = %d and it was %.3f and total train size is %d num attributes is %d key index is %d\n",idx, dist, testNumInstances, num_attributes, keyIndex);
                    //printf("train num instances is %d\n", train_num_instances);
                }
                
                
                //printf("our distance was %.3f, num classes is %d\n", dist, num_classes);
                // Add to our candidates
                for(int c = 0; c < k; c++){
                    if(dist < candidates[2*c]) {
                        // Found a new candidate
                        // Shift previous candidates down by one
                        for(int x = k-2; x >= c; x--) {
                            candidates[2*x+2] = candidates[2*x];
                            candidates[2*x+3] = candidates[2*x+1];
                        }
                        
                        // Set key vector as potential k NN
                        candidates[2*c ] = dist;
                        candidates[2*c+1 ] = train_matrix[keyIndex*num_attributes + num_attributes - 1]; // class value

                        break;
                    }
                }
            }

            // Bincount the candidate labels and pick the most common
            for(int i = 0; i < k; i++) {
                classCounts[(int)candidates[2*i+1 ] ] += 1;
                //printf("class counts here are %d\n", classCounts[(int)candidates[2*i+1 + candidatesOffset] + classCountsOffset]);
            }
            
            int max_value = -1;
            int max_class = 0;
            for(int i = 0; i < num_classes; i++) {
                if(classCounts[i] > max_value) {
                    max_value = classCounts[i ];
                    max_class = i;
                    if(idx < 10){
                        //printf("max class at 41 was %d and the classCountOfset is %d \n", max_class, classCountsOffset);
                        //printf("my thread is is %d\n", threadIdx.x);
                    }
                    //printf("class count at i= %d is %d\n", i, classCounts[i + classCountsOffset]);
                }
                
            }

            //printf("Thread %d: max_class = %d\n", idx, max_class);
            //printf("predictions %d is %d\n", idx, predictions[idx]);
            __syncthreads();
            
            
            
            predictionsGlobal[idx] = max_class;
        
            // Make prediction with 
            
            
            //printf("max class was %d\n, and predictions at %d was %d\n", max_class, idx, predictions[idx]);
    

            
            
            
            //printf("max class was %d\n, and predictionsGlobal at %d was %d\n", max_class, idx, predictionsGlobal[idx]);


        }
    }

    // Implements a threaded kNN where for each candidate query an in-place priority queue is maintained to identify the nearest neighbors
    int* KNN(ArffData* train, ArffData* test, int k, int num_threads) {     
        
        const char* inDataFilePath = "sortedDataBase.txt";

        FILE* file = fopen(inDataFilePath, "r");

        // Get the file size
        fseek(file, 0, SEEK_END);
        size_t file_size = ftell(file);
        rewind(file);
        

            // Allocate memory to hold the file contents
        char* h_text = (char*)malloc(file_size);

        // Read the file into the host buffer
        fread(h_text, 1, file_size, file);
        fclose(file);

        // Allocate memory on the GPU
        char* d_text;
        hipMalloc((void**)&d_text, file_size);

        // Copy the file contents to the GPU
        hipMemcpy(d_text, h_text, file_size, hipMemcpyHostToDevice);

        

        //so essentially my goal is to produce all of these itemssets 
        //i want to take a previous gpu solution and use NVIDIA's hardware based 
        //dynamic programming APIs 
        //we send the transactional database to the GPU
        //

    
        // int* predictions = (int*)malloc(test->num_instances() * sizeof(int));

        // int num_classes = train->num_classes();
        // int num_attributes = train->num_attributes();
        // int train_num_instances = train->num_instances();

        // int test_num_instances = test->num_instances();
        // int testNumClasses = test->num_classes();
        // int testNumAttributes = test-> num_attributes();
        

        // float *d_testSet;
        // float *d_trainSet;

        // int *d_predictions;

        // int numElements = num_attributes * train_num_instances;
        // int testNumElements = testNumAttributes * test_num_instances;
        // // Pointers representing the dataset as a 2D matrix size num_instances x num_attributes
        // float *train_matrix = train->get_dataset_matrix();  // Use directly
        // float *test_matrix = test->get_dataset_matrix();    // Use directly

        
        // int numStreams = 4;

        // //need to cudamalloc to hold the training data
        // //CUDA MALLOC for the train array
        // hipMalloc(&d_trainSet, (numElements) * sizeof(float));
        // hipMalloc(&d_testSet,(testNumElements) *  sizeof(float));
        // hipMalloc(&d_predictions, (test_num_instances * sizeof(int)));

        // hipStream_t *streams = (hipStream_t*) malloc (numStreams * sizeof(hipStream_t));

        // for (int i = 0; i < numStreams; i++){
        //     hipStreamCreate(&streams[i]);
        // }

        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) {
        //     printf("CUDA error: %s\n", hipGetErrorString(err));
        // }

        // //int numberTrainElementsPerStream = (numElements + numStreams - 1) / numStreams;
        // int numberTestElementsPerStream = (testNumElements + numStreams - 1)/numStreams;
        // int numTestInstancesPerStream = (test_num_instances + numStreams - 1)/numStreams;
        
        
        
        // hipEvent_t start, stop;
        // hipEventCreate(&start);
        // hipEventCreate(&stop);
        // float milliseconds = 0;
        
        // int threadsPerBlockDim = 64;

        // // Calculate the size of shared memory needed
        // int sharedMemSize = 0;                  // For classCounts
        // printf("size of memory usage is %d\n", sharedMemSize);
        // int gridDimSize = (numTestInstancesPerStream + threadsPerBlockDim - 1) / threadsPerBlockDim;

        // hipEventRecord(start);
        // printf("before we call the kernel\n");
        // //hipMemcpy(d_trainSet, train_matrix, (numElements)*sizeof(float), hipMemcpyHostToDevice);
        // printf("size of numberTestElementsPerStream is %d\n", numberTestElementsPerStream);
        // printf("size of numberTestInstancePerStream is %d\n", numTestInstancesPerStream);
        // for (int i = 0; i < numStreams; i++)
	    // {
        //     //copying the train set to our device (GPU)
        //     //0.041824 ms each way for 2 things... so why is performance with streams so poor?
        //     hipMemcpyAsync(d_trainSet, train_matrix, (numElements)*sizeof(float),hipMemcpyHostToDevice, streams[i]);
        //     hipMemcpyAsync(&d_testSet[i*numberTestElementsPerStream], &test_matrix[i*numberTestElementsPerStream], numberTestElementsPerStream  *  sizeof(float), hipMemcpyHostToDevice, streams[i]);
        //     //hipMemcpyAsync(&d_predictions[i*numTestInstancesPerStream], &predictions[i*numTestInstancesPerStream], numTestInstancesPerStream * sizeof(int), hipMemcpyHostToDevice, streams[i]);

        //     printStuff<<<gridDimSize, threadsPerBlockDim, sharedMemSize, streams[i]>>>(d_testSet, d_trainSet, numberTestElementsPerStream, train_num_instances, k, num_attributes, num_classes, d_predictions, test_num_instances, i, numTestInstancesPerStream);
            
        //     hipMemcpyAsync(&predictions[i*numTestInstancesPerStream], &d_predictions[i*numTestInstancesPerStream], numTestInstancesPerStream * sizeof(int), hipMemcpyDeviceToHost, streams[i]);

        // }

        // hipDeviceSynchronize();  // Ensure kernel finishes THIS IS SO IMPORTANT 
        // //OTHERWISE THE PROGRAM WILL JUST END BEFORE THE KERNELS EVEN GET A CHANCE TO LAUNCH

        // hipEventRecord(stop);
	    // hipEventSynchronize(stop);
        // hipEventElapsedTime(&milliseconds, start, stop);
	    // printf("v1 GPU option time to sum the matrixes %f ms\n", milliseconds);

        // hipDeviceProp_t deviceProp;
        // hipGetDeviceProperties(&deviceProp, 0);

        // printf("GPU %d: %s\n", 0, deviceProp.name);
        
        

        // printf("after we call the kernel\n");
      
        // hipFree(d_trainSet);
        // hipFree(d_testSet);
        // free(streams);
        // hipFree(d_predictions);

        

        // return predictions;
    }

    int* computeConfusionMatrix(int* predictions, ArffData* dataset)
    {
        int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matrix size numberClasses x numberClasses
        
        for(int i = 0; i < dataset->num_instances(); i++) { // for each instance compare the true class and predicted class    
            int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
            int predictedClass = predictions[i];
            //printf("predictions[%d] = %d , true class was %d\n", i, predictions[i], trueClass);
            confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
        }
        
    
        return confusionMatrix;
    }

    float computeAccuracy(int* confusionMatrix, ArffData* dataset)
    {
        int successfulPredictions = 0;
        
        for(int i = 0; i < dataset->num_classes(); i++) {
            successfulPredictions += confusionMatrix[i*dataset->num_classes() + i]; // elements in the diagonal are correct predictions
        }
        
        return 100 * successfulPredictions / (float) dataset->num_instances();
    }

    int main(int argc, char *argv[])
    {
        if(argc != 5)
        {
            printf("Usage: ./program datasets/train.arff datasets/test.arff k num_threads");
            exit(0);
        }

        // k value for the k-nearest neighbors
        int k = strtol(argv[3], NULL, 10);
        int num_threads = strtol(argv[4], NULL, 10);

        // Open the datasets
        ArffParser parserTrain(argv[1]);
        ArffParser parserTest(argv[2]);
        ArffData *train = parserTrain.parse();
        ArffData *test = parserTest.parse();
        
        struct timespec start, end;
        int* predictions = NULL;
        
        // Initialize time measurement
        clock_gettime(CLOCK_MONOTONIC_RAW, &start);
        
        predictions = KNN(train, test, k, num_threads);
        
        // Stop time measurement
        clock_gettime(CLOCK_MONOTONIC_RAW, &end);

        // Compute the confusion matrix
        int* confusionMatrix = computeConfusionMatrix(predictions, test);
        // Calculate the accuracy
        float accuracy = computeAccuracy(confusionMatrix, test);

        uint64_t time_difference = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;

        printf("The %i-NN classifier for %lu test instances and %lu train instances required %llu ms CPU time for threaded with %d threads. Accuracy was %.2f\%\n", k, test->num_instances(), train->num_instances(), (long long unsigned int) time_difference, accuracy, num_threads);

        // Print the predictions array
        // printf("Predictions array after hipMemcpy:\n");
        // for (int i = 0; i < test->num_instances(); i++) {
        //     printf("predictions[%d] = %d\n", i, predictions[i]);
        // }

        free(predictions);
        free(confusionMatrix);
    }

    /*  // Example to print the test dataset
        float* test_matrix = test->get_dataset_matrix();
        for(int i = 0; i < test->num_instances(); i++) {
            for(int j = 0; j < test->num_attributes(); j++)
                printf("%.0f, ", test_matrix[i*test->num_attributes() + j]);
            printf("\n");
        }
    */